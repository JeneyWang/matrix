#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>

#include "../matrix/definitions.h"

__global__ void multi(int *A, int *B, int *C)
{
	int cvalue = 0;
	int col = blockIdx.x * blockDim.x + threadIdx.x;
	int row = blockIdx.y * blockDim.y + threadIdx.y;


	if (row > DIM || col > DIM) return;

	for (int e = 0; e < DIM; ++e){
		cvalue += A[row*DIM + e] * B[e*DIM + col];
	}
	C[row*DIM + col] = cvalue;
}

void matrixmulti(int A[][DIM],int B[][DIM],int C[][DIM]){
	int *dev_a, *dev_b, *dev_c;

	//allocate memory on global memory of gpu
	hipError_t err = hipMalloc((void**)&dev_a, ((DIM)*(DIM))*sizeof(int));
	printf("Cuda malloc A:%s \n", hipGetErrorString(err));
	err = hipMalloc((void**)&dev_b, ((DIM)*(DIM))*sizeof(int));
	printf("Cuda malloc B:%s \n", hipGetErrorString(err));
	err = hipMalloc((void**)&dev_c, ((DIM)*(DIM))*sizeof(int));
	printf("Cuda malloc C:%s \n", hipGetErrorString(err));


	//Copy array A and B on device allocated memory
	err = hipMemcpy(dev_a, A, ((DIM*DIM))*sizeof(int), hipMemcpyHostToDevice);
	printf("Cuda memcpy to device A:%s \n", hipGetErrorString(err));
	err = hipMemcpy(dev_b, B, ((DIM*DIM))*sizeof(int), hipMemcpyHostToDevice);
	printf("Cuda memcpy to device B:%s \n", hipGetErrorString(err));

	//two dimension threads
	dim3 dimBlock(BlockSize, BlockSize);
	dim3 dimGrid((DIM + dimBlock.x - 1) / dimBlock.x, (DIM + dimBlock.y - 1) / dimBlock.y);

	//call the kernel function multi
	multi << < dimGrid, dimBlock >> >(dev_a, dev_b, dev_c);

	//retrieve array C from device memory
	err = hipMemcpy(C, dev_c, ((DIM*DIM))*sizeof(int), hipMemcpyDeviceToHost);
	printf("Cuda memcpy to HOST C:%s \n", hipGetErrorString(err));

	for (int i = 0; i < DIM; i++){
		for (int j = 0; j < DIM; j++){
			printf("C(%d,%d) = %d \n", i, j, C[i][j]);
		}
	}

	//free the memory
	hipFree(dev_a);
	hipFree(dev_b);
	hipFree(dev_c);
}