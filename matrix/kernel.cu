#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <random>
#include <time.h>
#include <math.h>

#define DIM 20
#define BlockSize 32


__global__ void multi(int *A, int *B, int *C)
{
	int cvalue = 0;
	//int cwidth = blockDim.x*gridDim.x, awidth = blockDim.x*gridDim.x, bwidth = blockDim.x*gridDim.x;
	int col = blockIdx.x * blockDim.x + threadIdx.x;
	int row = blockIdx.y * blockDim.y + threadIdx.y;
	//int offset = iy*(blockDim.x*gridDim.x) + ix;

	if (row > DIM || col > DIM) return;

	for (int e = 0; e < DIM; ++e){
		cvalue += A[row*DIM + e] * B[e*DIM + col];
	}
	C[row*DIM + col] = cvalue;
}

int main()
{
	srand(time(0));
	int A[DIM][DIM], B[DIM][DIM], C[DIM][DIM];
	int *dev_a, *dev_b, *dev_c;

	//allocate memory on global memory of gpu
	hipError_t err = hipMalloc((void**)&dev_a, ((DIM)*(DIM))*sizeof(int));
	printf("Cuda malloc A:%s \n", hipGetErrorString(err));
	err = hipMalloc((void**)&dev_b, ((DIM)*(DIM))*sizeof(int));
	printf("Cuda malloc B:%s \n", hipGetErrorString(err));
	err = hipMalloc((void**)&dev_c, ((DIM)*(DIM))*sizeof(int));
	printf("Cuda malloc C:%s \n", hipGetErrorString(err));
	

	//populate array A and B
	for (int i = 0; i<DIM; i++) {
		for (int j = 0; j < DIM; j++){
			A[i][j] = rand()%100;
			B[i][j] = rand()%100;
			//printf("A(%d,%d) = %d \n", i, j, A[i][j]);
			//printf("B(%d,%d) = %d \n", i, j, B[i][j]);
		}
	}

	//Copy array A and B on device allocated memory
	err = hipMemcpy(dev_a, A, ((DIM*DIM))*sizeof(int), hipMemcpyHostToDevice);
	printf("Cuda memcpy to device A:%s \n", hipGetErrorString(err));
	err = hipMemcpy(dev_b, B, ((DIM*DIM))*sizeof(int), hipMemcpyHostToDevice);
	printf("Cuda memcpy to device B:%s \n", hipGetErrorString(err));

	//two dimension threads
	dim3 dimBlock(BlockSize, BlockSize);
	dim3 dimGrid((DIM + dimBlock.x - 1) / dimBlock.x, (DIM + dimBlock.y - 1) / dimBlock.y);
	
	//call the kernel function multi
	multi <<< dimGrid,dimBlock >> >(dev_a, dev_b, dev_c);

	//retrieve array C from device memory
	err = hipMemcpy(C, dev_c, ((DIM*DIM))*sizeof(int), hipMemcpyDeviceToHost);
	printf("Cuda memcpy to HOST C:%s \n", hipGetErrorString(err));

	for (int i = 0; i < DIM; i++){
		for (int j = 0; j < DIM; j++){
			printf("C(%d,%d) = %d \n", i, j, C[i][j]);
		}
	}

	//free the memory
	hipFree(dev_a);
	hipFree(dev_b);
	hipFree(dev_c);

    return 0;
}

